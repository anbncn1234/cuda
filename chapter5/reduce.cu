#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../common/common.h"
#include <iostream>

#define DIM 128

int recursiveReduce(int *data, int const size){
    if (size == 1) return data[0];
    int const stride = size /2;
    for (int i = 0; i < stride; i ++){
        data[i] += data[i + stride];
    }
    return recursiveReduce( data, stride);
}

__global__ void warmup( int *g_idata, int *g_odata, unsigned int n){
    unsigned int tid  = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    
    if (blockDim.x >= 1024 &&  tid < 512) idata[tid] += idata[tid+ 512];
    __syncthreads();
    if (blockDim.x >= 512 &&  tid < 256) idata[tid] += idata[tid+ 256];
    __syncthreads();
    if (blockDim.x >= 256 &&  tid < 128) idata[tid] += idata[tid+ 128];
    __syncthreads();
    if (blockDim.x >= 128 &&  tid < 64) idata[tid] += idata[tid+ 64];
    __syncthreads();

    if (tid < 32){
        volatile int *vmem  = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if  (tid == 0){ g_odata[blockIdx.x] = idata[0];}
}

__global__ void reduceGmem( int *g_idata, int *g_odata, unsigned int n){
    unsigned int tid  = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    
    if (blockDim.x >= 1024 &&  tid < 512) idata[tid] += idata[tid+ 512];
    __syncthreads();
    if (blockDim.x >= 512 &&  tid < 256) idata[tid] += idata[tid+ 256];
    __syncthreads();
    if (blockDim.x >= 256 &&  tid < 128) idata[tid] += idata[tid+ 128];
    __syncthreads();
    if (blockDim.x >= 128 &&  tid < 64) idata[tid] += idata[tid+ 64];
    __syncthreads();

    if (tid < 32){
        volatile int *vmem  = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if  (tid == 0){ g_odata[blockIdx.x] = idata[0];}
}

__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n){
    __shared__ int smem[DIM];

    unsigned int tid  = threadIdx.x;
    // convert global data pointer to local pointer
    int *idata = g_idata + blockIdx.x * blockDim.x;

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    //set to smem by each threads
    smem[tid] = idata[tid];
    __syncthreads();

    if (blockDim.x >= 1024 &&  tid < 512) smem[tid] += smem[tid+ 512];
    __syncthreads();
    if (blockDim.x >= 512 &&  tid < 256) smem[tid] += smem[tid+ 256];
    __syncthreads();
    if (blockDim.x >= 256 &&  tid < 128) smem[tid] += smem[tid+ 128];
    __syncthreads();
    if (blockDim.x >= 128 &&  tid < 64) smem[tid] += smem[tid+ 64];
    __syncthreads();

    if (tid < 32){
        volatile int *vsmem  = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    if  (tid == 0){ g_odata[blockIdx.x] = smem[0];}
}

__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n){
    __shared__ int smem[DIM];

    unsigned int tid  = threadIdx.x;
    // convert global data pointer to local pointer
    int *idata = g_idata + blockIdx.x * blockDim.x;

    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    
    //unrolling 4 blocks
    int tmpSum = 0;
    if (idx + 3 * blockDim.x <= n){
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        tmpSum = a1 + a2 + a3 + a4;
    }

    //set to smem by each threads
    smem[tid] = tmpSum;
    __syncthreads();

    if (blockDim.x >= 1024 &&  tid < 512) smem[tid] += smem[tid+ 512];
    __syncthreads();
    if (blockDim.x >= 512 &&  tid < 256) smem[tid] += smem[tid+ 256];
    __syncthreads();
    if (blockDim.x >= 256 &&  tid < 128) smem[tid] += smem[tid+ 128];
    __syncthreads();
    if (blockDim.x >= 128 &&  tid < 64) smem[tid] += smem[tid+ 64];
    __syncthreads();

    if (tid < 32){
        volatile int *vsmem  = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    if  (tid == 0){ g_odata[blockIdx.x] = smem[0];}
}


__global__ void reduceSmemUnrollDyn(int *g_idata, int *g_odata, unsigned int n){
    extern __shared__ int smem[];

    unsigned int tid  = threadIdx.x;
    // convert global data pointer to local pointer
    int *idata = g_idata + blockIdx.x * blockDim.x;

    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    
    //unrolling 4 blocks
    int tmpSum = 0;
    if (idx + 3 * blockDim.x <= n){
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        tmpSum = a1 + a2 + a3 + a4;
    }

    //set to smem by each threads
    smem[tid] = tmpSum;
    __syncthreads();

    if (blockDim.x >= 1024 &&  tid < 512) smem[tid] += smem[tid+ 512];
    __syncthreads();
    if (blockDim.x >= 512 &&  tid < 256) smem[tid] += smem[tid+ 256];
    __syncthreads();
    if (blockDim.x >= 256 &&  tid < 128) smem[tid] += smem[tid+ 128];
    __syncthreads();
    if (blockDim.x >= 128 &&  tid < 64) smem[tid] += smem[tid+ 64];
    __syncthreads();

    if (tid < 32){
        volatile int *vsmem  = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    if  (tid == 0){ g_odata[blockIdx.x] = smem[0];}
}


int main(int argc , char **argv)
{
    printf("%s starting\n", argv[0]);

    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceprop;
    CHECK(hipGetDeviceProperties(&deviceprop,dev));
    printf("Using Device %d : %s\n", dev, deviceprop.name);
    
    int size = 1 << 24;
    int blocksize = 512;
    
    if (argc > 1){
        blocksize = atoi(argv[1]);
    }

    dim3 block(DIM, 1);  // 1d
    dim3 grid ((size + block.x - 1) / block.x, 1);
    
    size_t nBytes = size  * sizeof(int);
    int * h_idata = (int*) malloc(nBytes);
    int * h_odata = (int*) malloc( grid.x * sizeof(int));  //you duoshao ge block
    int * temp = (int*) malloc(nBytes);

    //initial the array
    for (int i = 0 ; i < size;i++){
        h_idata[i] = (int)(rand() & 0xff);
    }

    int sum = 0;
    for (int i = 0 ; i < size;i++){
        sum += h_idata[i];
    }
    printf("sum value is : %d\n", sum);

    memcpy(temp, h_idata, nBytes);

    int gpu_sum = 0;

    int *d_idata = NULL;
    int *d_odata = NULL;

    hipMalloc((void**)&d_idata, nBytes);
    hipMalloc((void**)&d_odata, grid.x * sizeof(int));

    
    //cpu sum
    Timer timer;
    timer.start();

    int cpu_sum = recursiveReduce(temp, size);
    timer.stop();
    float elapsedTime = timer.elapsedms();

    printf("cpu reduce time: %f,  sum: %d\n", elapsedTime, cpu_sum);

    //gpu sum
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    timer.start();
    warmup<<<grid.x, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize(); 
    timer.stop();
    float elapsedTime1 = timer.elapsedms();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i ++){
        gpu_sum += h_odata[i];
    }
    printf("warm up reduce time: %f,  sum: %d\n", elapsedTime1, gpu_sum);


    //gpu sum
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    timer.start();
    reduceGmem<<<grid.x, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize(); 
    timer.stop();
    elapsedTime1 = timer.elapsedms();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x ; i ++){
        gpu_sum += h_odata[i];
    }
    printf("reduceGmem gpu reduce time: %f,  sum: %d, gird ,block (%d %d)\n", elapsedTime1, gpu_sum, grid.x , block.x);

    //gpu sum
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    timer.start();
    reduceSmem<<<grid.x, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize(); 
    timer.stop();
    elapsedTime1 = timer.elapsedms();
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x ; i ++){
        gpu_sum += h_odata[i];
    }
    printf("reduceSmem gpu reduce time: %f,  sum: %d, gird ,block (%d %d)\n", elapsedTime1, gpu_sum, grid.x , block.x);

    //gpu sum
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    timer.start();
    reduceSmemUnroll<<<grid.x /4 , block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize(); 
    timer.stop();
    elapsedTime1 = timer.elapsedms();
    hipMemcpy(h_odata, d_odata, grid.x /4 * sizeof(int),hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 4 ; i ++){
        gpu_sum += h_odata[i];
    }
    printf("reduceSmemUnroll gpu reduce time: %f,  sum: %d, gird ,block (%d %d)\n", elapsedTime1, gpu_sum, grid.x / 4, block.x);

    //gpu sum
    hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    timer.start();
    reduceSmemUnrollDyn<<<grid.x /4 , block, DIM * sizeof(int)>>>(d_idata, d_odata, size);
    hipDeviceSynchronize(); 
    timer.stop();
    elapsedTime1 = timer.elapsedms();
    hipMemcpy(h_odata, d_odata, grid.x /4 * sizeof(int),hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 4 ; i ++){
        gpu_sum += h_odata[i];
    }
    printf("reduceSmemUnrollDyn gpu reduce time: %f,  sum: %d, gird ,block (%d %d)\n", elapsedTime1, gpu_sum, grid.x / 4, block.x);

    hipFree(d_idata);
    hipFree(d_odata);
    hipDeviceReset();

    free(h_idata);
    free(h_odata);
    free(temp);

    return 0;
}