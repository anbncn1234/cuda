#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

// 检查 CUDA 错误
#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        std::cerr << "CUDA Error: " << hipGetErrorString(status) << std::endl; \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

// 检查 cuSPARSE 错误
#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        std::cerr << "cuSPARSE Error: " << status << std::endl;                \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

int main() {
    // 初始化稠密矩阵 (4x4)
    const int m = 4; // 行数
    const int n = 4; // 列数
    const int nnz = 9; // 非零元素个数
    double h_A[m * n] = {
        1, 0, 0, 2,
        0, 3, 4, 0,
        5, 0, 6, 0,
        0, 7, 0, 8
    };

    // 打印稠密矩阵
    std::cout << "Dense Matrix:" << std::endl;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::cout << h_A[i * n + j] << " ";
        }
        std::cout << std::endl;
    }

    // 初始化 cuSPARSE
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // 分配设备内存并拷贝稠密矩阵到设备
    double *d_A;
    CHECK_CUDA(hipMalloc((void**)&d_A, m * n * sizeof(double)));
    CHECK_CUDA(hipMemcpy(d_A, h_A, m * n * sizeof(double), hipMemcpyHostToDevice));

    // 创建稠密矩阵描述符
    hipsparseDnMatDescr_t matA;
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matA, m, n, n, d_A, HIP_R_64F, HIPSPARSE_ORDER_ROW));

    // 创建稀疏矩阵描述符 (CSR 格式)
    hipsparseSpMatDescr_t matB;
    int *d_csrRowPtr, *d_csrColInd;
    double *d_csrVal;
    CHECK_CUDA(hipMalloc((void**)&d_csrRowPtr, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_csrColInd, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_csrVal, nnz * sizeof(double)));
    CHECK_CUSPARSE(hipsparseCreateCsr(&matB, m, n, nnz, d_csrRowPtr, d_csrColInd, d_csrVal,
                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // 分配缓冲区
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize));
    void *d_buffer;
    CHECK_CUDA(hipMalloc(&d_buffer, bufferSize));

    // 执行稠密矩阵转稀疏矩阵
    CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, d_buffer));
    CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, d_buffer));

    // 将结果拷贝回主机
    int h_csrRowPtr[m + 1];
    int h_csrColInd[nnz];
    double h_csrVal[nnz];
    CHECK_CUDA(hipMemcpy(h_csrRowPtr, d_csrRowPtr, (m + 1) * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_csrColInd, d_csrColInd, nnz * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_csrVal, d_csrVal, nnz * sizeof(double), hipMemcpyDeviceToHost));

    // 打印 CSR 格式的结果
    std::cout << "\nCSR RowPtr:" << std::endl;
    for (int i = 0; i <= m; i++) {
        std::cout << h_csrRowPtr[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "CSR ColInd:" << std::endl;
    for (int i = 0; i < nnz; i++) {
        std::cout << h_csrColInd[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "CSR Val:" << std::endl;
    for (int i = 0; i < nnz; i++) {
        std::cout << h_csrVal[i] << " ";
    }
    std::cout << std::endl;

    // 释放设备内存
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_csrRowPtr));
    CHECK_CUDA(hipFree(d_csrColInd));
    CHECK_CUDA(hipFree(d_csrVal));
    CHECK_CUDA(hipFree(d_buffer));

    // 销毁描述符和句柄
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matA));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matB));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}