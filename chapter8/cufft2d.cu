#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include "../common/common.h"
#include <iostream>

int main() {
    // 定义 FFT 变换的相关参数
    int nx = 4; // 输入数据的行数
    int ny = 4; // 输入数据的列数
    int size = sizeof(hipfftComplex) * nx * ny;
 
    // 定义输入和输出数组
    hipfftComplex *data;
    hipfftComplex *result;
 
    // 分配内存
    hipMalloc((void**)&data, size);
    hipMalloc((void**)&result, size);
 
    // 初始化输入数据（假设为一些复数值）
    // 这里只是示例，实际数据可以根据需求进行初始化
    hipfftComplex *hostData = (hipfftComplex*)malloc(size);
    for (int i = 0; i < nx; ++i) {
        for (int j = 0; j < ny; ++j) {
            int index = i * ny + j;
            hostData[index].x = i + j;  // 实部
            hostData[index].y = 0;      // 虚部
        }
    }
 
    // 将输入数据从主机内存拷贝到 GPU 内存中
    hipMemcpy(data, hostData, size, hipMemcpyHostToDevice);
 
    // 创建 cuFFT 计划
    hipfftHandle plan;
    hipfftPlan2d(&plan, nx, ny, HIPFFT_C2C);
 
    // 执行 FFT 变换
    hipfftExecC2C(plan, data, result, HIPFFT_FORWARD);
 
    // 释放 cuFFT 计划
    hipfftDestroy(plan);
 
    // 将结果从 GPU 内存拷贝回主机内存
    hipMemcpy(hostData, result, size, hipMemcpyDeviceToHost);
 
    // 打印输出结果
    printf("FFT Result:\n");
    for (int i = 0; i < nx; ++i) {
        for (int j = 0; j < ny; ++j) {
            int index = i * ny + j;
            printf("(%f, %f)\t", hostData[index].x, hostData[index].y);
        }
        printf("\n");
    }
 
    // 释放内存
    hipFree(data);
    hipFree(result);
    free(hostData);
 
    return 0;
}