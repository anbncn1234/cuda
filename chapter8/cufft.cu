#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include "../common/common.h"
#include <iostream>

int main() {
    // 定义 FFT 变换的相关参数
    int n = 8; // 输入数据的大小
    int size = sizeof(hipfftComplex) * n;
 
    // 定义输入和输出数组
    hipfftComplex *data;
    hipfftComplex *result;
 
    // 分配内存
    hipMalloc((void**)&data, size);
    hipMalloc((void**)&result, size);
 
    // 初始化输入数据（假设为一些复数值）
    // 这里只是示例，实际数据可以根据需求进行初始化
    hipfftComplex *hostData = (hipfftComplex*)malloc(size);
    for (int i = 0; i < n; ++i) {
        hostData[i].x = i;  // 实部
        hostData[i].y = 0;  // 虚部
    }
 
    // 将输入数据从主机内存拷贝到 GPU 内存中
    hipMemcpy(data, hostData, size, hipMemcpyHostToDevice);
 
    // 创建 cuFFT 计划
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
 
    // 执行 FFT 变换
    hipfftExecC2C(plan, data, result, HIPFFT_FORWARD);
 
    // 释放 cuFFT 计划
    hipfftDestroy(plan);
 
    // 将结果从 GPU 内存拷贝回主机内存
    hipMemcpy(hostData, result, size, hipMemcpyDeviceToHost);
 
    // 打印输出结果
    printf("FFT Result:\n");
    for (int i = 0; i < n; ++i) {
        printf("(%f, %f)\n", hostData[i].x, hostData[i].y);
    }
 
    // 释放内存
    hipFree(data);
    hipFree(result);
    free(hostData);
 
    return 0;
}