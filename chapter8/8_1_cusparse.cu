#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../common/common.h"
#include <iostream>

int main(void){
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceprop;
    CHECK(hipGetDeviceProperties(&deviceprop,dev));
    printf("device %d: %s \n", dev, deviceprop.name);

    hipStream_t *handle = (hipStream_t *)malloc(sizeof(hipStream_t));
    //create cusparse handle
    hipsparseCreate(&handle)

    /*
    tensor([[1, 7, 0, 0],
        [0, 2, 8, 0],
        [5, 0, 3, 9],
        [0, 6, 0, 4]])
    >>> sp.csr()
    (row_ptr = tensor([0, 2, 4, 7, 9]), 
    col_ind = tensor([0, 1, 1, 2, 0, 2, 3, 1, 3]), 
    values = tensor([1, 7, 2, 8, 5, 3, 9, 6, 4]))

     */
    int n_vals = 9;
    int n_rows = 4;
    int n_cols = 4;

    float *h_csrVals;
    int *h_csrCols;
    int *h_csrRows;

    h_csrVals[9] =  [1.0, 7.0, 2.0, 8.0, 5.0, 3.0, 9.0, 6.0, 4.0];
    h_csrCols[9] = [0, 1, 1, 2, 0, 2, 3, 1, 3];
    h_csrRows[n_rows + 1] = [0, 2, 4, 7, 9];


    float *d_csrVals;
    int *d_csrCols;
    int *d_csrRows; 
    
    hipMalloc((void **)&d_csrVals, n_vals * sizeof(float));
    hipMalloc((void **)&d_csrCols, n_vals * sizeof(int));
    hipMalloc((void **)&d_csrRows, (n_rows + 1) * sizeof(int));

    hipMemcpy(d_csrVals, h_csrVals, n_vals* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_csrCols, h_csrCols, n_vals* sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrRows, h_csrRows, (n_rows + 1)* sizeof(int), hipMemcpyHostToDevice);

    
    free(h_csrVals);
    free(h_csrRows);
    free(h_csrCols);
    hipFree(d_csrVals);
    hipFree(d_csrCols);
    hipFree(d_csrRows);

    hipDeviceReset();

    return 0;
}