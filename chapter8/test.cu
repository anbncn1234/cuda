
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <hipblas.h>
 
int main() {
    std::cout << "cuSPARSE version: " << CUSPARSE_VERSION / 1000 << "." << (CUSPARSE_VERSION % 1000) / 100 << "." << (CUSPARSE_VERSION % 100) << std::endl;
    std::cout << "cublas version: " << CUBLAS_VERSION / 1000 << "." << (CUBLAS_VERSION % 1000) / 100 << "." << (CUBLAS_VERSION % 100) << std::endl;
    return 0;
}