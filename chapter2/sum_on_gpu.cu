#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <windows.h>

#define CHECK(call) \
 {\
    const hipError_t error = call; \
    if (error != hipSuccess)\
    {\
        printf("Error: %s: %d\n", __FILE__, __LINE__);\
        printf("code :%d reason :%s\n", error , hipGetErrorString(error));\
        exit(1);\
    }\
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i])> epsilon)
        {
            match = 0;
            printf("Array do not match\n");
            printf("host %5.2f gpu % 5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;

        }
    }
    if (match) printf("array matches\n");
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx< N; idx ++)
    {
        C[idx] = A[idx] + B[idx];
    }
}

void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned int) time(&t));

    for (int i = 0; i < size; i++) {
        ip[i] = (float) (rand() & 0xff) / 10.0f;
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C)
{
    int i  = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

/*
double cpusec()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double)tp.tv_usec* 1.e-6);
}
*/

int main(int argc , char **argv)
{
    printf("%s starting\n", argv[0]);

    int dev = 0;
    hipSetDevice(dev);


    //set up data
    int nElem = 32;
    size_t nBytes = nElem * sizeof(float);
    float *h_A, *h_B, *hostRef, *gpuRef;

    h_A = (float *) malloc (nBytes);
    h_B = (float *) malloc (nBytes);
    hostRef = (float *) malloc (nBytes);
    gpuRef = (float *) malloc (nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef,0, nBytes);
    memset(gpuRef,0, nBytes);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    //transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    dim3 block(nElem);
    dim3 grid(nElem/block.x);

    sumArraysOnGPU<<<grid,block>>>(d_A, d_B, d_C);
    printf("execution config <<<%d, %d>>>\n", grid.x, block.x);

    //copy kernel result back to host
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    checkResult(hostRef, gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}