
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx< N; idx ++)
    {
        C[idx] = A[idx] + B[idx];
    }
}

void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned int) time(&t));

    for (int i = 0; i < size; i++) {
        ip[i] = (float) (rand() & 0xff) / 10.0f;
    }
}

int main(int argc , char **argv)
{
    int nElem = 1024;
    size_t nBytes = nElem * sizeof(float);
    float *h_A, *h_B, *h_C;

    h_A = (float *) malloc (nBytes);
    h_B = (float *) malloc (nBytes);
    h_C = (float *) malloc (nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    sumArraysOnHost(h_A, h_B, h_C, nElem);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}