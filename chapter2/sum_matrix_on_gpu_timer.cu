#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <windows.h>

typedef unsigned long DWORD;

#define CHECK(call) \
 {\
    const hipError_t error = call; \
    if (error != hipSuccess)\
    {\
        printf("Error: %s: %d\n", __FILE__, __LINE__);\
        printf("code :%d reason :%s\n", error , hipGetErrorString(error));\
        exit(1);\
    }\
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i])> epsilon)
        {
            match = 0;
            printf("Array do not match\n");
            printf("host %5.2f gpu % 5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;

        }
    }
    if (match) printf("array matches\n");
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx< N; idx ++)
    {
        C[idx] = A[idx] + B[idx];
    }
}

void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned int) time(&t));

    for (int i = 0; i < size; i++) {
        ip[i] = (float) (rand() & 0xff) / 10.0f;
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C)
{
    int i  = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] + B[i];
}


void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny){
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix =0; ix < nx; ix++){
            ic[ix] = ia[ix] + ib[ix];
        }
        ia += nx; 
        ib += nx;
        ic += nx;
    }
}

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy*nx + ix;

    if (ix < nx && iy < ny){
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}


int main(int argc , char **argv)
{
    printf("%s starting\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceprop;
    CHECK(hipGetDeviceProperties(&deviceprop,dev));
    printf("Using Device %d : %s\n", dev, deviceprop.name);
    CHECK(hipSetDevice(dev));

    //set up data
    int nx  = 1<<14;
    int ny  = 1<<14;
    int nxy = nx * ny;
    size_t nBytes = nxy  * sizeof(float);
    printf("matrix size %d %d\n", nx, ny);

    float *h_A, *h_B, *hostRef, *gpuRef;

    h_A = (float *) malloc (nBytes);
    h_B = (float *) malloc (nBytes);
    hostRef = (float *) malloc (nBytes);
    gpuRef = (float *) malloc (nBytes);

    initialData(h_A, nxy);
    initialData(h_B, nxy);

    memset(hostRef,0, nBytes);
    memset(gpuRef,0, nBytes);

    // malloc device global memory
    float *d_MatA, *d_MatB, *d_MatC;
    hipMalloc((float**)&d_MatA, nBytes);
    hipMalloc((float**)&d_MatB, nBytes);
    hipMalloc((float**)&d_MatC, nBytes);

    //transfer data from host to device
    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);


    int dimx = 16;
    int dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1)/block.x, (ny + block.y - 1)/block.y);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);

    sumMatrixOnGPU2D<<<grid,block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    hipDeviceSynchronize();
    

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("execution config <<<(%d,%d), (%d,%d)>>>\n", grid.x,grid.y, block.x, block.y);
    printf("Kernel execution time: %f ms\n", milliseconds);



    hipEventDestroy(start);
    hipEventDestroy(stop);

    
    //copy kernel result back to host
    hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost);

    sumMatrixOnHost(h_A, h_B, hostRef, nx,ny);

    checkResult(hostRef, gpuRef, nxy);

    hipFree(d_MatA);
    hipFree(d_MatB);
    hipFree(d_MatC);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}