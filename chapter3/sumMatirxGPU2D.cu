#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <windows.h>
#include "../common/common.h"

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i])> epsilon)
        {
            match = 0;
            printf("Array do not match\n");
            printf("host %5.2f gpu % 5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;

        }
    }
    if (match) printf("array matches\n");
}

void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned int) time(&t));

    for (int i = 0; i < size; i++) {
        ip[i] = (float) (rand() & 0xff) / 10.0f;
    }
}



void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny){
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix =0; ix < nx; ix++){
            ic[ix] = ia[ix] + ib[ix];
        }
        ia += nx; 
        ib += nx;
        ic += nx;
    }
}

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy*nx + ix;

    if (ix < nx && iy < ny){
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}


int main(int argc , char **argv)
{
    printf("%s starting\n", argv[0]);



    int dev = 0;
    hipDeviceProp_t deviceprop;
    CHECK(hipGetDeviceProperties(&deviceprop,dev));
    printf("Using Device %d : %s\n", dev, deviceprop.name);
    
    int nx = 1 << 14;
    int ny = 1<< 14;
    int nxy = nx * ny;
    int dimx = 32, dimy =32;
    if (argc > 2){
        dimx = atoi(argv[1]);
        dimy = atoi(argv[2]);
    }


    size_t nBytes = nxy  * sizeof(float);
    //printf("matrix size %d %d\n", nx, ny);

    float *h_A, *h_B, *hostRef, *gpuRef;


    h_A = (float *) malloc (nBytes);
    h_B = (float *) malloc (nBytes);
    hostRef = (float *) malloc (nBytes);
    gpuRef = (float *) malloc (nBytes);

    initialData(h_A, nxy);
    initialData(h_B, nxy);

    memset(hostRef,0, nBytes);
    memset(gpuRef,0, nBytes);

    // malloc device global memory
    float *d_MatA, *d_MatB, *d_MatC;
    hipMalloc((float**)&d_MatA, nBytes);
    hipMalloc((float**)&d_MatB, nBytes);
    hipMalloc((float**)&d_MatC, nBytes);

    //transfer data from host to device
    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1)/block.x,(ny + block.y - 1)/block.y);
    printf("execution config:  grid (%d %d), block (%d %d)  ", block.x, block.y, grid.x, grid.y);
    
    

    Timer timer;
    timer.start();
    hipDeviceSynchronize();
    sumMatrixOnGPU2D<<<grid,block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    hipDeviceSynchronize();
    timer.stop();
    float elapsedTime = timer.elapsedms();
    
    printf("execution time: %f ms\n", elapsedTime);
    
    //copy kernel result back to host
    hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost);

    sumMatrixOnHost(h_A, h_B, hostRef, nx,ny);

    checkResult(hostRef, gpuRef, nxy);

    hipFree(d_MatA);
    hipFree(d_MatB);
    hipFree(d_MatC);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);


    return 0;
}