#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <windows.h>

typedef unsigned long DWORD;

#define CHECK(call) \
 {\
    const hipError_t error = call; \
    if (error != hipSuccess)\
    {\
        printf("Error: %s: %d\n", __FILE__, __LINE__);\
        printf("code :%d reason :%s\n", error , hipGetErrorString(error));\
        exit(1);\
    }\
}

__global__ void mathKernel1( float *C){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;
    if (tid % 2 == 0){
        a = 100.0f;
    }
    else{
        b = 200.0f;
    }
    C[tid] = a + b;
}

__global__ void mathKernel2( float *C){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;
    if ((tid / warpSize) % 2 == 0){
        a = 100.0f;
    }
    else{
        b = 200.0f;
    }
    C[tid] = a + b;
}

__global__ void mathKernel3( float *C){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;
    bool ipred = (tid % 2 == 0);
    if (ipred){
        a = 100.0f;
    }
    if  (!ipred){
        b = 200.0f;
    }
    C[tid] = a + b;
}

__global__ void warmingup( float *C){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;
    if ((tid / warpSize) % 2 == 0){
        a = 100.0f;
    }
    else{
        b = 200.0f;
    }
    C[tid] = a + b;
}


int main(int argc , char **argv)
{
    printf("%s starting\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceprop;
    CHECK(hipGetDeviceProperties(&deviceprop,dev));
    printf("Using Device %d : %s\n", dev, deviceprop.name);
    
    int size = 64;
    int blocksize = 64;
    if (argc > 1) blocksize = atoi(argv[1]);
    if (argc > 2) size      = atoi(argv[2]);
    printf("Data size %d\n", size);


    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1)/block.x);
    printf("execution config: %d %d\n", block.x, grid.x);
    
    float *d_C;
    size_t nBytes = size * sizeof(float);
    hipMalloc((float**) &d_C, nBytes);
    

    LARGE_INTEGER frequency;
    LARGE_INTEGER start;
    LARGE_INTEGER end;
 
    QueryPerformanceFrequency(&frequency);
    QueryPerformanceCounter(&start);
 
    // 执行你想要计时的代码
    // ...
    hipDeviceSynchronize();
    warmingup<<<grid,block>>>(d_C);
    hipDeviceSynchronize();
 
    QueryPerformanceCounter(&end);
 
    double time = (double)(end.QuadPart - start.QuadPart) / (double)frequency.QuadPart * 1.0E6;
    printf("execute time: %f ms\n", time);

/*
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("warmup execution time: %f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    */
    // kernel 1

    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    
    hipEventRecord(start1);
    hipDeviceSynchronize();
    mathKernel1<<<grid,block>>>(d_C);
    hipDeviceSynchronize();
    
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    
    float milliseconds1 = 0;
    hipEventElapsedTime(&milliseconds1, start1, stop1);
    
    printf("Kernel1 execution time: %f ms\n", milliseconds1);
    hipEventDestroy(start1);
    hipEventDestroy(stop1);

    // kernel 2

    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    
    hipEventRecord(start2);
    hipDeviceSynchronize();
    mathKernel2<<<grid,block>>>(d_C);
    hipDeviceSynchronize();
    
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    
    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start2, stop2);
    
    printf("Kernel2 execution time: %f ms\n", milliseconds2);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    // kernel 3

    hipEvent_t start3, stop3;
    hipEventCreate(&start3);
    hipEventCreate(&stop3);
    
    hipEventRecord(start3);
    hipDeviceSynchronize();
    mathKernel3<<<grid,block>>>(d_C);
    hipDeviceSynchronize();
    
    hipEventRecord(stop3);
    hipEventSynchronize(stop3);
    
    float milliseconds3 = 0;
    hipEventElapsedTime(&milliseconds3, start3, stop3);
    
    printf("Kernel3 execution time: %f ms\n", milliseconds3);
    hipEventDestroy(start3);
    hipEventDestroy(stop3);

    hipFree(d_C);
    hipDeviceReset();

    return 0;
}